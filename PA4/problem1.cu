#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <omp.h>
#include <iostream>

#include <k_means.cu>

#define h  800 
#define w  800

#define input_file  "input.raw"
#define output_file "output.raw"

int main(int argc, char** argv){
    int i;
    FILE *fp;

  	unsigned char *a = (unsigned char*) malloc (sizeof(unsigned char)*h*w);
    
	// the matrix is stored in a linear array in row major fashion
	if (!(fp=fopen(input_file, "rb"))) {
		printf("can not opern file\n");
		return 1;
	}
	fread(a, sizeof(unsigned char), w*h, fp);
	fclose(fp);
    
	// MY CODE STARTS HERE -------
	// measure the start time here
	clock_t start,end;
	start = clock();

	int size = h * w;

	// Define means
	unsigned char cluster1_mean = 0;
	unsigned char cluster2_mean = 85;
	unsigned char cluster3_mean = 170;
	unsigned char cluster4_mean = 255;

	unsigned char *clusters;
	hipMallocManaged(&clusters, size*sizeof(unsigned char));

	// Initialize all clusters to 0
	for (int i = 0; i < size; i++) {
		clusters[i] = 0;
	}

	// Run 30 iterations of k-means
	for (int i = 0; i < 30; i++) {
		// Need 800x800 = 640000 threads, CUDA GPUs run kernels using blocks of threads that are a multiple of 32 and no greater than 1024 (Compute Compatability 2.x and later) and 512 for 1.x
		// Choosing 1250 blocks and 512 threads per block - Source: https://stackoverflow.com/questions/9985912/how-do-i-choose-grid-and-block-dimensions-for-cuda-kernels
		k_means<<<1250,512>>>(a, clusters, cluster1_mean, cluster2_mean, cluster3_mean, cluster4_mean);

		hipDeviceSynchronize();

		int c1_total, c2_total, c3_total, c4_total = 0;
		unsigned char c1_vals, c2_vals, c3_vals, c4_vals = 0;

		for (int i = 0; i < size; i++) {
			if (clusters[i] == 1) {
				c1_total += a[i];
				c1_vals++;
			} else if (clusters[i] == 2) {
				c2_total += a[i];
				c2_vals++;
			} else if (clusters[i] == 3) {
				c3_total += a[i];
				c3_vals++;
			} else if (clusters[i] == 4) {
				c4_total += a[i];
				c4_vals++;
			}
		}

		cluster1_mean = c1_total/c1_vals;
		cluster2_mean = c2_total/c2_vals;
		cluster3_mean = c3_total/c3_vals;
		cluster4_mean = c4_total/c4_vals;
		std::cout << "Means for iteration " << i << ": c1=" << cluster1_mean << " c2= " << cluster2_mean << " c3= " << cluster3_mean << " c4= " << cluster4_mean << std::endl;
	}	

	// Write to output.raw
	if (!(fp=fopen(output_file,"wb"))) {
		printf("can not opern file\n");
		return 1;
	}	
	fwrite(a, sizeof(unsigned char),w*h, fp);
    fclose(fp);

	// Free Mem
    hipFree(clusters);
	
	// measure the end time here
    end = clock();
    double timeElapsed = ((double)((end-start)))/(double)(CLOCKS_PER_SEC);

	// print out the execution time here
    std::cout << "Time: " << timeElapsed << " seconds" << std::endl;
    
    return 0;
}